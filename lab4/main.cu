#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hipblas.h> // подключение библиотеки cuBLAS
#define IDX2C(i,j,ld) (((j)*(ld))+(i)) // макрос для работы с индексами в стиле FORTRAN
int main() {
const int N = 6;
hipblasHandle_t handle;
float *dev_A, *dev_b;
float *x, *A, *b;
x = (float *)malloc(N * sizeof(*x));
b = (float *)malloc(N * sizeof(*b));
A = (float *)malloc(N * N * sizeof(*A));

int ind = 11; // инициализация матрицы и вектора правой части
for (int j = 0; j < N; j++) {
for (int i = 0; i < N; i++)
if (i >= j)
A[IDX2C(i, j, N)] = (float)ind++;
else A[IDX2C(i, j, N)] = 0.0f;
b[j] = 1.0f; }
// выделяем память на GPU соответствующего размера для каждой переменной
hipMalloc((void**)&dev_b, N * sizeof(*x));
hipMalloc((void**)&dev_A, N * N * sizeof(*A));
hipblasCreate(&handle); // инициализируем контекст cuBLAS
// копируем вектор и матрицу из CPU в GPU
hipblasSetVector(N, sizeof(*b), b, 1, dev_b, 1);
hipblasSetMatrix(N, N, sizeof(*A), A, N, dev_A, N);

// решаем нижнюю треугольню матрицу
hipblasStrsv(handle, HIPBLAS_FILL_MODE_LOWER,
HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, N, dev_A, N, dev_b, 1);
// копируем результат из GPU в CPU
hipblasGetVector(N, sizeof(*x), dev_b, 1, x, 1);
for (int i = 0; i < N; i++) {
for (int j = 0; j < N; j++)
printf("%3.0f ", A[IDX2C(i, j, N)]);
printf(" = %f %4.6f\n", b[i], x[i]); }
hipFree(dev_b); // освобождаем память в GPU
hipFree(dev_A);
hipblasDestroy(handle); // уничтожаем контекст cuBLAS
free(x); free(b); free(A); // освобождаем память в CPU
}
