#include<stdlib.h>
#include<stdio.h>
#include<string.h>
#include<math.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

typedef float2 Complex;
#define N_ELEM 10
int
main(int argc, char **argv)
{
    Complex *h_signal;
    Complex *d_signal;
    int mem_size;
    hipfftHandle plan;

    printf("[truely simpleCUFFT] is starting...\n");
    //Выделяем память для входных данных на хосте
    h_signal = (Complex *)malloc(sizeof(Complex) * N_ELEM);

    //Инициализируем входные данные
    for (unsigned int i = 0; i < N_ELEM; ++i){
        h_signal[i].x = (float)i;
        h_signal[i].y = 0;
        printf("%f\t%f\n", h_signal[i].x, h_signal[i].y);
    }

    //Выделяем память для входных данных на видеокарте и копируем их туда
    mem_size = sizeof(Complex) * N_ELEM;
    hipMalloc((void **)&d_signal, mem_size);
    hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice);

    //Собственно выполняем преобразование Фурье
    printf("Transforming signal cufftExecC2C\n");
    hipfftPlan1d(&plan, N_ELEM, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);

    //Копируем результат вычисления с видеокарты на хост
    hipMemcpy(h_signal, d_signal, mem_size, hipMemcpyDeviceToHost);
    for (unsigned int i = 0; i < N_ELEM; ++i)
        printf("%f\t%f\n", h_signal[i].x, h_signal[i].y);

    //Прибираем за собой
    hipfftDestroy(plan);
    free(h_signal);
    hipFree(d_signal);
    hipDeviceReset();

    exit(0);
}